#define _BSD_SOURCE

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <time.h>
#include <sys/time.h>

#include <iostream>
#include <cstdio>

using namespace std;

#define cudaSucceeded(ans) { cudaAssert((ans), __FILE__, __LINE__); }

inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) {
    std::cerr << "cudaAssert failed: "
              << hipGetErrorString(code)
              << file << ":" << line
              << std::endl;
    if (abort) {
      exit(code);
    }
  }
}

int useconds() {
  struct timeval t;
  gettimeofday(&t, NULL);
  return t.tv_sec*1000000+t.tv_usec;
}

int main() {
  for (int i = 1; i < 1<<30; i *= 2) {
    int *mem;
    const int num_tries = 100;
    int time = 0;
    int min_time = 0x7FFFFFFF;
    int max_time = 0;
    for (int j = 0; j < num_tries; j++) {
      int start = useconds();
      cudaSucceeded(hipMalloc(&mem, i));
      cudaSucceeded(hipMemset(mem, 0, 0));
      cudaSucceeded(hipDeviceSynchronize());
      int aft = useconds();
      int this_time = aft-start;
      time += this_time;
      min_time = min_time < this_time ? min_time : this_time;
      max_time = max_time < this_time ? this_time : max_time;
      cudaSucceeded(hipFree(mem));
    }
    printf("%d bytes; average: %dus; min: %dus; max: %dus\n", i, time/num_tries, min_time, max_time);
  }
}

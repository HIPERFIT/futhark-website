#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform_scan.h>
#include <iostream>
#include <cstdlib>
#include <sys/time.h>

#include "common.hpp"

typedef thrust::tuple<int, int, int, int> MSSTuple;

class prepare : public thrust::unary_function<const int, MSSTuple> {
public:
  __host__ __device__ MSSTuple operator()(const int x) const {
    return MSSTuple(max(x,0), max(x,0), max(x,0), x);
  }
};

class combine : public thrust::binary_function<const MSSTuple &, const MSSTuple &, MSSTuple>  {
public:
  __host__ __device__ MSSTuple operator()(const MSSTuple &x,
                                          const MSSTuple &y) const {
    int mssx = thrust::get<0>(x);
    int misx = thrust::get<1>(x);
    int mcsx = thrust::get<2>(x);
    int tsx  = thrust::get<3>(x);

    int mssy = thrust::get<0>(y);
    int misy = thrust::get<1>(y);
    int mcsy = thrust::get<2>(y);
    int tsy  = thrust::get<3>(y);

    return MSSTuple(max(mssx, max(mssy, mcsx + misy)),
                    max(misx, tsx+misy),
                    max(mcsy, mcsx+tsy),
                    tsx + tsy);
  }
};

int main(int argc, char **argv) {
  int runs, n;
  runs_and_n(argc, argv, &runs, &n);

  thrust::device_vector<int> d(n);
  thrust::device_vector<int> dres(n);

  init_vector(&d);

  // Warmup
  thrust::transform_inclusive_scan
      (d.begin(), d.end(),
       thrust::make_zip_iterator(thrust::make_tuple
                                 (dres.begin(),
                                  thrust::make_discard_iterator(),
                                  thrust::make_discard_iterator(),
                                  thrust::make_discard_iterator())),
       prepare(),
       combine());

  start_timing();
  for (size_t i = 0; i < runs; ++i) {
    thrust::transform_inclusive_scan
      (d.begin(), d.end(),
       thrust::make_zip_iterator(thrust::make_tuple
                                 (dres.begin(),
                                  thrust::make_discard_iterator(),
                                  thrust::make_discard_iterator(),
                                  thrust::make_discard_iterator())),
       prepare(),
       combine());
  }
  end_timing();

  std::cout << "Result: " << dres[n-1] << std::endl;
  report_time(runs);
}
